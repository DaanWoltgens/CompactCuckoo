#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <iterator>
#include <set>
#include <inttypes.h>
#include <atomic>
#include <random>
#include "SpinBarrier.h"

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

//For to List
#include <vector>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#ifndef MAIN
#define MAIN
#include "main.h"
#endif

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#ifndef SHAREDQUEUE
#define SHAREDQUEUE
#include "SharedQueue.cu"
#endif


//Taken from Better GPUs
template <int tile_sz>
struct bucket {
    // Constructor to load the key-value pair of the bucket 2
    GPUHEADER
    bucket(ClearyCuckooEntry<addtype, remtype>* ptr, cg::thread_block_tile<tile_sz> tile, int bucketIndex, int bucketSize) : ptr_(ptr), tile_(tile) {
        tIndex = bucketIndex * bucketSize + tile_.thread_rank();
        lane_pair_ = ptr[tIndex];
    }

    // Compute the load of the bucket
    GPUHEADER
    int compute_load() {
        auto load_bitmap = tile_.ballot((lane_pair_.getO()));
        return __popc(load_bitmap);
    }

    // Find the value associated with a key
    GPUHEADER
    bool find(const remtype rem) {
        bool key_exist = (rem == lane_pair_.getR());
        int key_lane = __ffs(tile_.ballot(key_exist));
        if (key_lane == 0) return false;
        return tile_.shfl(true, key_lane - 1);
    }

    // Perform an exchange operation
    GPUHEADER_D
    ClearyCuckooEntry<addtype, remtype> exch_at_location(ClearyCuckooEntry<addtype, remtype> pair, const int loc) {
        ClearyCuckooEntry<addtype, remtype> old_pair;
        printf("%i: \t\t\t\tExch in bucket: thread_rank %i loc:%i\n", getThreadID(), tile_.thread_rank(), loc);
        if (tile_.thread_rank() == loc) {
            printf("%i: \t\t\t\tActual Exch\n", getThreadID());
            ptr_[tIndex].exchValue(&pair);
        }
        printf("%i: \t\t\t\t Exch Done\n", getThreadID());
        return tile_.shfl(pair, loc);
    }

    private:
        ClearyCuckooEntry<addtype, remtype>* ptr_;
        ClearyCuckooEntry<addtype, remtype> lane_pair_;
        const cg::thread_block_tile<tile_sz>  tile_;
        int tIndex = 0;
};

template <int tile_sz>
class ClearyCuckooBucketed: HashTable{

/*
*
*  Global Variables
*
*/
    private:
        //Constant Vars
        const static int HS = 59;       //HashSize
        int MAXLOOPS = 25;
        int MAXREHASHES = 30;

        //Vars at Construction
        const int RS;                         //RemainderSize
        const int AS;                         //AdressSize
        const int B;                          //NumBuckets
        const int Bs = tile_sz;               //BucketSize
        
        int tablesize;
        int occupancy = 0;

        //Hash tables
        ClearyCuckooEntry<addtype, remtype>* T;

        int hashcounter = 0;

        //Hash function ID
        int hn;
        int* hashlist;

        //Bucket Variables
        int* bucketIndex;

        //Flags
#ifdef GPUCODE
        int failFlag = 0;
        int occupation = 0;
        int rehashFlag = 0;
#else
        std::atomic<int> failFlag;
        std::atomic<int> occupation;
        std::atomic<int> rehashFlag;
#endif
        SharedQueue<keytype>* rehashQueue;

        //Method to init the hashlsit
        GPUHEADER
        void createHashList(int* list) {
            for (int i = 0; i < hn; i++) {
                list[i] = i;
            }
            return;
        }

        //Method to iterate over hashes (Rehashing)
        GPUHEADER
        void iterateHashList(int* list) {
            ////printf("\tUpdating Hashlist\n");
            for (int i = 0; i < hn; i++) {
                list[i] = (list[i]+1+i)%32;
            }
            return;
        }

        //Method to select next hash to use in insertion
        GPUHEADER
        int getNextHash(int* ls, int curr) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == curr) {
                    if (i + 1 != hn) {
                        return ls[i + 1];
                    }
                    else {
                        return ls[0];
                    }
                }
            }

            //Default return 0 if hash can't be found
            return ls[0];
        }

        //Checks if hash ID is contained in hashlist
        GPUHEADER
        bool containsHash(int* ls, int query) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == query) {
                    return true;
                }
            }
            return false;
        }

#ifdef GPUCODE
        //Method to set Flags on GPU(Failure/Rehash)
        GPUHEADER_D
        bool setFlag(int* loc, int val, bool strict=true) {
            int val_i = val == 0 ? 1 : 0;

            //In devices, atomically exchange
            uint64_cu res = atomicCAS(loc, val_i, val);
            //Make sure the value hasn't changed in the meantime
            if ( (res != val_i) && strict) {
                return false;
            }
            __threadfence();
            return true;
        }

#else
        GPUHEADER
        //Method to set Flags on CPU (Failure/Rehash)
        bool setFlag(std::atomic<int>* loc, int val, bool strict=true) {
            int val_i = val == 0 ? 1 : 0;
                ////printf("%i:\t:Attempting CAS\n", getThreadID());
            if (std::atomic_compare_exchange_strong(loc, &val_i, val)) {
                ////printf("%i:\t:Flag Set\n", getThreadID());
                return true;
            }else{
              return false;
            }
        }
#endif


        /**
         * Internal Insertion Loop
         **/
        GPUHEADER_D
        bool insertIntoTable(keytype k, ClearyCuckooEntry<addtype, remtype>* T, int* hs, cg::thread_block_tile<tile_sz> tile, int depth=0){
            printf("%i: \t\tInsert into Table\n", getThreadID());
            keytype x = k;
            int hash = hs[0];

            //If the key is already inserted don't do anything
            printf("%i: \t\t\tLookup\n", getThreadID());
            if (lookup(k, T, tile)) {
                return false;
            }

            //Start the iteration
            int c = 0;

            printf("%i: \t\t\tEntering Loop\n", getThreadID());
            while (c < MAXLOOPS) {
                //Get the add/rem of k
                hashtype hashed1 = RHASH(hash, x);
                addtype add = getAdd(hashed1, AS);
                remtype rem = getRem(hashed1, AS);

                auto cur_bucket = bucket<tile_sz>(T, tile, add, Bs);
                auto load = cur_bucket.compute_load();

                printf("%i: \t\t\tLoad - %i\n", getThreadID(), load);

                addtype bAdd;

                if (load == Bs) {
                    bAdd = RHASH(0, add) % Bs; //select some location within the table
                }
                else {
                    bAdd = load;
                }

                ClearyCuckooEntry<addtype, remtype> entry(rem, hash, true, false);
                entry = cur_bucket.exch_at_location(entry, bAdd);

                
                //Store the old value
                remtype temp = entry.getR();
                bool wasoccupied = entry.getO();
                int oldhash = entry.getH();

                printf("%i: \t\t\told: rem:%" PRIu64 " Occ:%i hash:%i \n", getThreadID(), temp, wasoccupied, oldhash);

                //If the old val was empty return
                if (!wasoccupied) {
                    return true;
                }

                //Otherwise rebuild the original key
                hashtype h_old = reformKey(add, temp, AS);
                x = RHASH_INVERSE(oldhash, h_old);
                

                //Hash with the next hash value
                hash = getNextHash(hs, oldhash);

                c++;
            }
        };


        //Method to check for duplicates after insertions
        GPUHEADER
        void removeDuplicates(keytype k) {
            //To store whether value was already encountered
            bool found = false;

            for (int i = 0; i < hn; i++) {
                uint64_cu hashed1 = RHASH(hashlist[i], k);
                addtype add = getAdd(hashed1, AS);
                remtype rem = getRem(hashed1, AS);
                for (int j = 0; j < Bs; j++) {

                    if (T[add*Bs +j].getH() == hashlist[i] && T[add*Bs + j].getR() == rem && T[add*Bs +j].getO()) {
                        //If value was already found
                        if (found) {
                            //Mark as not occupied
                            T[add*Bs + j].setO(false);
                        }
                        //Mark value as found
                        found = true;
                    }
                }
            }
        }

        //Lookup internal method
        GPUHEADER
        bool lookup(uint64_cu k, ClearyCuckooEntry<addtype, remtype>* T, cg::thread_block_tile<tile_sz>& tile){
            for (int i = 0; i < hn; i++) {
                uint64_cu hashed1 = RHASH(hashlist[i], k);
                addtype add = getAdd(hashed1, AS);
                remtype rem = getRem(hashed1, AS);
                
                auto cur_bucket = bucket<tile_sz>(T, tile, add , Bs);
                auto res = cur_bucket.find(rem);

                return res;
            }
        };

        GPUHEADER
        void print(ClearyCuckooEntry<addtype, remtype>* T) {
            printf("----------------------------------------------------------------\n");
            printf("|    i     |     R[i]       | O[i] |        key         |label |\n");
            printf("----------------------------------------------------------------\n");
            printf("Tablesize %i\n", tablesize);
            
            for (int i = 0; i < B; i++) {
                printf("----------------------------------------------------------------\n");
                printf("|                   Bucket %i                                   \n", i);
                printf("----------------------------------------------------------------\n");
                for (int j = 0; j < Bs; j++) {
                    remtype rem = T[i*Bs + j].getR();
                    int label = T[i*Bs + j].getH();
                    hashtype h = reformKey(i, rem, AS);
                    keytype k = RHASH_INVERSE(label, h);

                    printf("|%-10i|%-16" PRIu64 "|%-6i|%-20" PRIu64 "|%-6i|\n", j, T[i*Bs + j].getR(), T[i*Bs + j].getO(), k, T[i*Bs + j].getH());
                }
            }

            printf("------------------------------------------------------------\n");
        }


    public:
        /**
         * Constructor
         */
        ClearyCuckooBucketed() : ClearyCuckooBucketed(4,1,1){}

        ClearyCuckooBucketed(int addressSize, int hashNumber) : 
            AS(addressSize), RS(HS - AS), B( (int)pow(2, AS) ){
            //printf("Constructor\n");
            tablesize = B*Bs;
            
            int queueSize = std::max(100, (int)(tablesize / 10));

            hn = hashNumber;

            //Allocating Memory for tables
            //printf("\tAlloc Mem\n");
#ifdef GPUCODE
            gpuErrchk(hipMallocManaged(&T, tablesize * sizeof(ClearyCuckooEntry<addtype, remtype>)));
            gpuErrchk(hipMallocManaged(&hashlist, hn * sizeof(int)));
            gpuErrchk(hipMallocManaged((void**)&rehashQueue, sizeof(SharedQueue<int>)));
            gpuErrchk(hipMallocManaged(&bucketIndex, Bs * sizeof(int)));
#else
            T = new ClearyCuckooEntry<addtype, remtype>[numBuckets*Bs + bucketSize];
            hashlist = new int[hn];
            bucketIndex = new int[Bs];
#endif
            //printf("\tInit Entries\n");
            //Init table entries
            for(int i=0; i<B; i++){
                for (int j = 0; j < Bs; j++) {
                    //printf("\t\tEntry %i %i\n",i, j);
                    new (&T[i*Bs + j]) ClearyCuckooEntry<addtype, remtype>();
                }
                bucketIndex[i] = 0;
            }
            //printf("\tCreate Hashlist\n");
            //Create HashList
            createHashList(hashlist);
            //printf("\tInit Complete\n");
        }

        /**
         * Destructor
         */
        ~ClearyCuckooBucketed(){
            //printf("Destructor\n");
            #ifdef GPUCODE
            gpuErrchk(hipFree(T));
            gpuErrchk(hipFree(hashlist));
            gpuErrchk(hipFree(bucketIndex));

            #else
            delete[] T;
            delete[] hashlist;
            delete[] bucketIndex;
            #endif
        }

        //Taken from Better GPU Hash Tables
        GPUHEADER_D
        bool coopInsert(bool to_insert, keytype k) {
            printf("%i: \tcoopInsert %" PRIu64"\n", getThreadID(), k);
            cg::thread_block thb = cg::this_thread_block();
            auto tile = cg::tiled_partition<tile_sz>(thb);
            printf("%i: \tTiledPartition\n", getThreadID());
            auto thread_rank = tile.thread_rank();
            bool success = true;
            //Perform the insertions
            uint32_t work_queue;
            while (work_queue = tile.ballot(to_insert)) {
                printf("%i: \tThread Starting Insertion\n", getThreadID());
                auto cur_lane = __ffs(work_queue) - 1;
                auto cur_k = tile.shfl(k, cur_lane);
                auto cur_result = insertIntoTable(cur_k, T, hashlist, tile);
                printf("%i: \tthread_rank:%i  cur_lane:%i\n", getThreadID(), tile.thread_rank(), cur_lane);
                if (tile.thread_rank() == cur_lane) {
                    to_insert = false;
                    success = cur_result;
                }
                printf("%i: \tInsertion Done\n", getThreadID());
                if (tile.thread_rank() == 0) {
                    print();
                }
            }
            return success;
        }

        //Public insertion call
        GPUHEADER_D
#ifdef GPUCODE
            bool insert(uint64_cu k) {
#else
            bool insert(uint64_cu k, SpinBarrier * barrier) {
#endif
            printf("%i:Insert %" PRIu64 "\n", getThreadID(), k);

            //Stores success/failure of rehash
            bool finalRes = false;
            if (coopInsert(true, k)) {
                //Reset the Hash Counter

                finalRes = true;
            }

            //Duplicate Check Phase
#ifdef DUPCHECK
#ifdef GPUCODE
            __syncthreads();
#else
            barrier->Wait();
#endif
            //Do duplicate Check if insertion was successful
            if (finalRes) {
                removeDuplicates(k);
            }

#ifdef GPUCODE
            __syncthreads();
#else
            barrier->Wait();
#endif
#endif

            return finalRes;
        };

        //Public Lookup call
        GPUHEADER_D
        bool coopLookup(bool to_lookup, uint64_cu k){
            printf("%i: Coop Lookup\n", getThreadID());
            //Iterate over hash functions and check if found
            cg::thread_block thb = cg::this_thread_block();
            cg::thread_block_tile<tile_sz> tile = cg::tiled_partition<tile_sz>(thb);
            auto thread_rank = tile.thread_rank();
            bool success = true;
            //Perform the insertions

            uint32_t work_queue;
            while (work_queue = tile.ballot(to_lookup)) {
                auto cur_lane = __ffs(work_queue) - 1;
                auto cur_k = tile.shfl(k, cur_lane);
                auto cur_result = lookup(cur_k, T, tile);
                
                if (tile.thread_rank() == cur_lane) {
                    to_lookup = false;
                    success = cur_result;
                }
            }
            return success;
            //printf("\t\t Lookup Failed\n");
            return false;
        };

        //Clear all Table Entries
        GPUHEADER
        void clear(){
            for (int i = 0; i < B; i++) {
                for (int j = 0; j < Bs; j++) {
                    new (&T[i*Bs + j]) ClearyCuckooEntry<addtype, remtype>();
                }
            }
        }

        //Get the size of the Table
        GPUHEADER
        int getSize(){
            return tablesize;
        }

        //Return a copy of the hashlist
        GPUHEADER
        int* getHashlistCopy() {
            int* res = new int[hn];
            for (int i = 0; i < hn; i++) {
                res[i] = hashlist[i];
            }
            return res;
        }

        //Transform a vector to a list
        GPUHEADER_H
        std::vector<uint64_cu> toList() {
            std::vector<uint64_cu> list;
            for (int i = 0; i < tablesize; i++) {
                for (int j = 0; j < tablesize; j++) {
                    if (T[i * Bs + j].getO()) {
                        hashtype h_old = reformKey(i, T[i * Bs + j].getR(), AS);
                        keytype x = RHASH_INVERSE(T[i * Bs + j].getH(), h_old);
                        list.push_back(x);
                    }
                }
            }
            return list;
        }

        //Iterate through all entries and do a read
        void readEverything(int N) {
            int j = 0;
            int step = 1;

            if (N < tablesize) {
                step = std::ceil(((float)tablesize) / ((float)N));
            }

            for (int i = 0; i < N; i+=step) {
                for (int k = 0; k < Bs; k++) {
                    j += T[(i * Bs + j) % tablesize].getR();
                }
            }

            if (j != 0) {
                //printf("Not all Zero\n");
            }
        }


        //Public print call
        GPUHEADER
        void print(){
            //printf("Hashlist:");
            for (int i = 0; i < hn; i++) {
                //printf("%i,", hashlist[i]);
            }
            //printf("\n");
            print(T);
        }

        //Method used for debugging
        GPUHEADER
        void debug(uint64_cu i) {

        }

        //Set the number of rehashes allowed
        void setMaxRehashes(int x){
            MAXREHASHES = x;
        }

        //Set the number of loops allowed
        void setMaxLoops(int x){
            MAXLOOPS = x;
        }

        //Get the number of hashes
        int getHashNum() {
            return hn;
        }

        int getBucketSize() {
            return Bs;
        }

};

//Method to fill ClearyCuckooBucketedtable
template <int tile_sz>
GPUHEADER_G
#ifdef GPUCODE
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, int* failFlag=nullptr, addtype begin = 0, int id = 0, int s = 1)
#else
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, SpinBarrier* barrier, int* failFlag = nullptr, addtype begin = 0, int id = 0, int s = 1)
#endif
{
#ifdef GPUCODE
    int index = threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif
    ////printf("Thread %i Starting\n", getThreadID());
    for (int i = index + begin; i < N + begin; i += stride) {
#ifdef GPUCODE
        if (!(H->insert(vals[i]))) {
#else
        if (!(H->insert(vals[i]))) {
#endif
            if (failFlag != nullptr) {
                (*failFlag) = true;
            }
            break;
        }

        //H->print();
    }
    ////printf("Insertions %i Over\n", getThreadID());
#ifdef DUPCHECK
#ifndef GPUCODE
    barrier->signalThreadStop();
#endif
#endif
}


#ifdef GPUCODE
//Method to fill ClearyCuckooBucketedtable with a failCheck on every insertion
template <int tile_sz>
GPUHEADER_G
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz> * H, addtype* occupancy, int* failFlag, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        if (failFlag[0]) {
            break;
        }
#ifdef GPUCODE
        if (!(H->insert(vals[i]))) {
#else
        if (!(H->insert(vals[i]))) {
#endif
            atomicCAS(&(failFlag[0]), 0, 1);
            break;
        }
        atomicAdd(&occupancy[0], 1);
    }
}
#endif

#ifndef GPUCODE
//Method to fill ClearyCuckooBucketed table with a failCheck on every insertion
GPUHEADER_G
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz> * H, SpinBarrier* barrier, std::atomic<addtype>* occupancy, std::atomic<bool>* failFlag, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        if ((*failFlag).load()) {
            break;
        }
#ifdef GPUCODE
        if (!(H->insert(vals[i]))) {
#else
        if (!(H->insert(vals[i]))) {
#endif
            (*failFlag).store(true);
            break;
        }
        (*occupancy).fetch_add(1);
    }
#ifdef DUPCHECK
#ifndef GPUCODE
    barrier->signalThreadStop();
#endif
#endif
}
#endif

//Method to check whether a ClearyCuckooBucketed table contains a set of values
template <int tile_sz>
GPUHEADER_G
void checkClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, bool* res, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        if ( !(H->coopLookup(true, vals[i])) ) {
            res[0] = false;
        }
    }
}

//Method to do lookups in a ClearyCuckooBucketed table on an array of values
template <int tile_sz>
GPUHEADER_G
void lookupClearyCuckooBucketed(int N, int start, int end, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, int id = 0, int s = 1) {
#ifdef GPUCODE
    int index = threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        H->coopLookup(true, vals[(i + start) % end]);
    }
}


