#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <iterator>
#include <set>
#include <inttypes.h>
#include <atomic>
#include <random>
#include "SpinBarrier.h"

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

//For to List
#include <vector>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#ifndef MAIN
#define MAIN
#include "main.h"
#endif

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#ifndef SHAREDQUEUE
#define SHAREDQUEUE
#include "SharedQueue.cu"
#endif


//Taken from Better GPUs
template <int tile_sz>
struct bucket {
    // Constructor to load the key-value pair of the bucket 2
    GPUHEADER
        bucket(ClearyCuckooEntryCompact<addtype, remtype>* ptr, cg::thread_block_tile<tile_sz> tile, int add) : ptr_(ptr), tile_(tile) {

        tIndex = (addtype)add;
        lane_pair_ = ptr[tIndex];
        subIndex = tile_.thread_rank();
    }

    // Compute the load of the bucket
    GPUHEADER
        int compute_load() {
        auto load_bitmap = tile_.ballot((ptr_[tIndex].getO(subIndex)));
        //printf("\t\t\t\tLoadBitmap: %i\n", load_bitmap);
        return __popc(load_bitmap);
    }

    // Find the value associated with a key
    GPUHEADER_D
        bool find(const remtype rem, const int hID) {
        //TODO
        bool key_exist = ((rem == ptr_[tIndex].getR(subIndex)) && (ptr_[tIndex].getO(subIndex)) && (ptr_[tIndex].getH(subIndex) == hID));
        //printf("%i:\tKey_exist %i\n", getThreadID(), key_exist);
        int key_lane = __ffs(tile_.ballot(key_exist));
        if (key_lane == 0) return false;
        return tile_.shfl(true, key_lane - 1);
    }

    // Find the value associated with a key
    GPUHEADER_D
    void removeDuplicates(const remtype rem, const int hID, bool* found) {
        //Check if val in loc is key
        bool key_exists = ((rem == ptr_[tIndex].getR(subIndex)) && (ptr_[tIndex].getO(subIndex)) && (ptr_[tIndex].getH(subIndex) == hID));
        //printf("%i:\tKey_exist at %i %i (%" PRIu64 " == %" PRIu64 ") %i (%i == %i))\n", getThreadID(), tIndex, key_exists, ptr_[tIndex].getR(), rem, ptr_[tIndex].getO(), ptr_[tIndex].getH(), hID);
        //printf("%i: i:%i O(i):%i R(i):%" PRIu64 "\n", getThreadID(), tIndex, ptr_[tIndex].getO(), ptr_[tIndex].getR());

        int realAdd = -1;
        //If first group where val is encountered, keep the first entry
        int num_vals = __popc(tile_.ballot(key_exists));
        int first = __ffs(tile_.ballot(key_exists)) - 1;
        //printf("NumVals:%i First:%i\n", num_vals, first);

        if ( (num_vals > 0) && !(*found) ) {
            //Mark as found for next iteration
            (*found) = true;
            realAdd = first;
            //printf("%i:\tRealAdd %i\n", getThreadID(), realAdd);
        }

        //If duplicate, mark as empty
        if (key_exists && (tile_.thread_rank() != realAdd)) {
            ptr_[tIndex].setO(false, subIndex);
        }


        return;
    }

    // Perform an exchange operation
    GPUHEADER_D
        uint64_cu exch_at_location(ClearyCuckooEntryCompact<addtype, remtype> pair, const int loc) {
        ClearyCuckooEntryCompact<addtype, remtype> old_pair;
        //printf("%i: \t\t\t\tExch in bucket: thread_rank %i loc:%i\n", getThreadID(), tile_.thread_rank(), loc);
        if (tile_.thread_rank() == loc) {
            //printf("%i: \t\t\t\tActual Exch Table:%" PRIu64 "  New:%" PRIu64 "\n", getThreadID(), ptr_[tIndex].getValue(), pair.getValue());
            ptr_[tIndex].tableSwap(&pair, subIndex, 0);
            //printf("%i: \t\t\t\tExch Done %" PRIu64 " Old %" PRIu64 "\n", getThreadID(), ptr_[tIndex].getValue(), pair.getValue());
        }
        //printf("%i: \t\t\t\tExch Return %" PRIu64 " %i \n", getThreadID(), pair.getValue(), loc);
        return tile_.shfl(pair.getValue(), loc);
    }

    private:
        ClearyCuckooEntryCompact<addtype, remtype>* ptr_;
        ClearyCuckooEntryCompact<addtype, remtype> lane_pair_;
        const cg::thread_block_tile<tile_sz>  tile_;
        int tIndex = 0;
        int subIndex = 0;
};

template <int tile_sz>
class ClearyCuckooBucketed: HashTable{

/*
*
*  Global Variables
*
*/
    private:
        //Constant Vars
        const static int HS = 59;       //HashSize
        int MAXLOOPS = 25;
        int MAXREHASHES = 30;

        const int ENTRYSIZE2 = 64;

        //Vars at Construction
        const int RS;                         //RemainderSize
        const int AS;                         //AdressSize
        const int B;                          //NumBuckets
        const int Bs = tile_sz;               //BucketSize

        int tablesize;
        int numEntries;

        int occupancy = 0;

        //Hash tables
        ClearyCuckooEntryCompact<addtype, remtype>* T;

        int hashcounter = 0;

        //Hash function ID
        int hn;
        int* hashlist;

        //Bucket Variables
        int* bucketIndex;

        //Flags
#ifdef GPUCODE
        int failFlag = 0;
        int occupation = 0;
        int rehashFlag = 0;
#else
        std::atomic<int> failFlag;
        std::atomic<int> occupation;
        std::atomic<int> rehashFlag;
#endif
        SharedQueue<keytype>* rehashQueue;

        //Method to init the hashlsit
        GPUHEADER
        void createHashList(int* list) {
            for (int i = 0; i < hn; i++) {
                list[i] = i;
            }
            return;
        }

        //Method to iterate over hashes (Rehashing)
        GPUHEADER
        void iterateHashList(int* list) {
            ////printf("\tUpdating Hashlist\n");
            for (int i = 0; i < hn; i++) {
                list[i] = (list[i]+1+i)%32;
            }
            return;
        }

        //Method to select next hash to use in insertion
        GPUHEADER
        int getNextHash(int* ls, int curr) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == curr) {
                    if (i + 1 != hn) {
                        return ls[i + 1];
                    }
                    else {
                        return ls[0];
                    }
                }
            }

            //Default return 0 if hash can't be found
            return ls[0];
        }

        //Checks if hash ID is contained in hashlist
        GPUHEADER
        bool containsHash(int* ls, int query) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == query) {
                    return true;
                }
            }
            return false;
        }

#ifdef GPUCODE
        //Method to set Flags on GPU(Failure/Rehash)
        GPUHEADER_D
        bool setFlag(int* loc, int val, bool strict=true) {
            int val_i = val == 0 ? 1 : 0;

            //In devices, atomically exchange
            uint64_cu res = atomicCAS(loc, val_i, val);
            //Make sure the value hasn't changed in the meantime
            if ( (res != val_i) && strict) {
                return false;
            }
            __threadfence();
            return true;
        }

#else
        GPUHEADER
        //Method to set Flags on CPU (Failure/Rehash)
        bool setFlag(std::atomic<int>* loc, int val, bool strict=true) {
            int val_i = val == 0 ? 1 : 0;
                ////printf("%i:\t:Attempting CAS\n", getThreadID());
            if (std::atomic_compare_exchange_strong(loc, &val_i, val)) {
                ////printf("%i:\t:Flag Set\n", getThreadID());
                return true;
            }else{
              return false;
            }
        }
#endif


        /**
         * Internal Insertion Loop
         **/
        GPUHEADER_D
            result insertIntoTable(keytype k, ClearyCuckooEntryCompact<addtype, remtype>* T, int* hs, cg::thread_block_tile<tile_sz> tile, int depth=0){
            //printf("%i: \t\tInsert into Table %" PRIu64 "\n", getThreadID(), k);
            keytype x = k;
            int hash = hs[0];

            //If the key is already inserted don't do anything
            //printf("%i: \t\t\tLookup\n", getThreadID());
            if (lookup(k, T, tile)) {
                return FOUND;
            }

            //Start the iteration
            int c = 0;

            //printf("%i: \t\t\tEntering Loop %i\n", getThreadID(), MAXLOOPS);
            while (c < MAXLOOPS) {
                //Get the add/rem of k
                hashtype hashed1 = RHASH(HFSIZE_BUCKET, hash, x);
                //printf("%i: \t\t\tHASHED %" PRIu64 "\n", getThreadID(), hashed1);
                addtype add = getAdd(hashed1, AS);
                remtype rem = getRem(hashed1, AS);

                auto cur_bucket = bucket<tile_sz>(T, tile, add);
                auto load = cur_bucket.compute_load();

                //printf("%i: \t\t\tLoad at %" PRIu32 " : %i\n", getThreadID(), add, load);

                addtype bAdd;

                if (load == Bs) {
                    bAdd = (addtype) (RHASH(HFSIZE_BUCKET, 0, rem) % Bs); //select some location within the table
                    //printf("%i: \t\t\tRandom Add at %" PRIu32 "\n", getThreadID(), bAdd);
                }
                else {
                    bAdd = load;
                }

                ClearyCuckooEntryCompact<addtype, remtype> entry(rem, hash, true, tile_sz, 0, false );
                //printf("%i: \t\t\tEntry %" PRIu64 "\n", getThreadID(), entry.getValue());
                ClearyCuckooEntryCompact<addtype, remtype> swapped(tile_sz, cur_bucket.exch_at_location(entry, bAdd));
                //printf("%i: \t\t\tEntryAFter %" PRIu64 "\n", getThreadID(), swapped.getValue());

                //Store the old value
                remtype temp = swapped.getR(0);
                bool wasoccupied = swapped.getO(0);
                int oldhash = swapped.getH(0);

                //printf("%i: \t\t\told: rem:%" PRIu64 " Occ:%i hash:%i \n", getThreadID(), temp, wasoccupied, oldhash);

                //If the old val was empty return
                if (!wasoccupied) {
                    //printf("%i: \t\tInsert Success\n", getThreadID());
                    return INSERTED;

                }

                //Otherwise rebuild the original key
                hashtype h_old = reformKey(add, temp, AS);
                keytype old_key = x;
                x = RHASH_INVERSE(HFSIZE_BUCKET, oldhash, h_old);
                if (old_key == x) {
                    return FOUND;
                }

                //printf("%i: \t\t\tRebuilt key:%" PRIu64 "\n", getThreadID(), x);


                //Hash with the next hash value
                hash = getNextHash(hs, oldhash);

                c++;
            }
            //printf("%i: \t\tInsert Fail\n", getThreadID());
            return FAILED;
        };


        //Method to check for duplicates after insertions
        GPUHEADER_D
        void removeDuplicates(keytype k, cg::thread_block_tile<tile_sz> tile) {
            //printf("%i: \t\tRemove Dups\n", getThreadID());
            //To store whether value was already encountered
            bool found = false;

            //Iterate over Hash Functions
            for (int i = 0; i < hn; i++) {
                uint64_cu hashed1 = RHASH(HFSIZE_BUCKET, hashlist[i], k);
                addtype add = getAdd(hashed1, AS);
                remtype rem = getRem(hashed1, AS);

                auto cur_bucket = bucket<tile_sz>(T, tile, add);
                cur_bucket.removeDuplicates(rem, hashlist[i], &found);
            }
            //printf("%i: \t\tDups Removed\n", getThreadID());
        }

        //Lookup internal method
        GPUHEADER_D
        bool lookup(uint64_cu k, ClearyCuckooEntryCompact<addtype, remtype>* T, cg::thread_block_tile<tile_sz> tile){
            //printf("%i: \t\tLookup\n", getThreadID());
            //Iterate over hash functions
            for (int i = 0; i < hn; i++) {
                uint64_cu hashed1 = RHASH(HFSIZE_BUCKET, hashlist[i], k);
                addtype add = getAdd(hashed1, AS);
                remtype rem = getRem(hashed1, AS);

                //printf("%i: Searching for %" PRIu64 " at %" PRIu32 "\n", getThreadID(), k, add);

                auto cur_bucket = bucket<tile_sz>(T, tile, add);
                auto res = cur_bucket.find(rem, hashlist[i]);
                if (res) {
                    //printf("%i: \t\tLookup Success\n", getThreadID());
                    return true;
                }
            }
            //printf("%i: \t\tLookup Fail\n", getThreadID());
            return false;
        };

        GPUHEADER
        void print(ClearyCuckooEntryCompact<addtype, remtype>* T) {
            printf("----------------------------------------------------------------\n");
            printf("|    i     |     R[i]       | O[i] |        key         |label |\n");
            printf("----------------------------------------------------------------\n");
            printf("Tablesize %i\n", tablesize);

            for (int i = 0; i < B; i++) {
                printf("----------------------------------------------------------------\n");
                printf("|                   Bucket %i                                   \n", i);
                printf("----------------------------------------------------------------\n");
                for (int j = 0; j < Bs; j++) {

                    int add = i * Bs + j;

                    addtype real_add = (addtype)(add / tile_sz);
                    addtype subIndex = (addtype)(add % tile_sz);


                    remtype rem = T[real_add].getR(subIndex);
                    int label = T[real_add].getH(subIndex);
                    hashtype h = reformKey(i, rem, AS);
                    keytype k = RHASH_INVERSE(HFSIZE_BUCKET, label, h);

                    printf("|%-10i|%-16" PRIu64 "|%-6i|%-20" PRIu64 "|%-6i|\n", j, T[real_add].getR(subIndex), T[real_add].getO(subIndex), k, T[real_add].getH(subIndex));
                }
            }

            printf("------------------------------------------------------------\n");
        }


    public:
        /**
         * Constructor
         */
        ClearyCuckooBucketed() : ClearyCuckooBucketed(4,1){}

        ClearyCuckooBucketed(int addressSize, int hashNumber) :
            AS( addressSize - ((int)log2(tile_sz))), B((int)pow(2, AS)), RS(HS - AS){
            //printf("Constructor\n");
            //printf("AS:%i tile_sz:%i, log2(tile_sz):%i", AS, tile_sz, (int) log2(tile_sz));

            tablesize = (B * Bs);
            numEntries = (int)(tablesize / tile_sz);

            int queueSize = std::max(100, (int)(tablesize / 10));

            hn = hashNumber;

            //Allocating Memory for tables
            //printf("\tAlloc Mem\n");
#ifdef GPUCODE
            gpuErrchk(hipMallocManaged(&T, (numEntries) * sizeof(ClearyCuckooEntryCompact<addtype, remtype>)));
            gpuErrchk(hipMallocManaged(&hashlist, hn * sizeof(int)));
            gpuErrchk(hipMallocManaged((void**)&rehashQueue, sizeof(SharedQueue<int>)));
#else
            T = new ClearyCuckooEntryCompact<addtype, remtype>[numEntries];
            hashlist = new int[hn];
#endif
            //printf("\tInit Entries\n");
            //Init table entries
            for(int i=0; i<numEntries; i++){
                    //printf("\t\tEntry %i %i\n",i, j);
                new (&T[i]) ClearyCuckooEntryCompact<addtype, remtype>(tile_sz);
            }

            //Default MAXLOOPS Value
            //1.82372633e+04 -2.60749645e+02  1.76799265e-02 -1.80594901e+04
            /*
            const double A = 18237.2633;
            const double x0 = -260.749645;
            const double k = .0176799265;
            const double off = -18059.4901;

            MAXLOOPS = std::max( MAXLOOPS, (int) ceil((A / (1.0 + exp(-k * (((double)AS) - x0)))) + off) );
            */
            //printf("\tCreate Hashlist\n");
            //Create HashList
            createHashList(hashlist);
            //printf("\tInit Complete\n");
        }

        /**
         * Destructor
         */
        ~ClearyCuckooBucketed(){
            //printf("Destructor\n");
            #ifdef GPUCODE
            gpuErrchk(hipFree(T));
            gpuErrchk(hipFree(hashlist));

            #else
            delete[] T;
            delete[] hashlist;
            #endif
        }

        //Taken from Better GPU Hash Tables
        GPUHEADER_D
        void coopDupCheck(bool to_check, keytype k) {
            //printf("%i: \tcoopInsert %" PRIu64"\n", getThreadID(), k);
            cg::thread_block thb = cg::this_thread_block();
            auto tile = cg::tiled_partition<tile_sz>(thb);
            //printf("%i: \tTiledPartition\n", getThreadID());
            auto thread_rank = tile.thread_rank();
            //Perform the insertions
            uint32_t work_queue;
            while (work_queue = tile.ballot(to_check)) {

                auto cur_lane = __ffs(work_queue) - 1;
                auto cur_k = tile.shfl(k, cur_lane);
                //printf("%i: \tThread Starting Insertion of %" PRIu64 "\n", getThreadID(), cur_k);
                removeDuplicates(cur_k, tile);
                if (tile.thread_rank() == cur_lane) {
                    to_check = false;
                }
                //printf("%i: \tInsertion Done\n", getThreadID());
            }
            //printf("%i: \tInsertion of  %" PRIu64" result:%i\n", getThreadID(), k, success);
            return;
        }

        //Taken from Better GPU Hash Tables
        GPUHEADER_D
        result coopInsert(bool to_insert, keytype k) {
            //printf("%i: \tcoopInsert %" PRIu64"\n", getThreadID(), k);
            cg::thread_block thb = cg::this_thread_block();
            auto tile = cg::tiled_partition<tile_sz>(thb);
            //printf("%i: \tTiledPartition\n", getThreadID());
            auto thread_rank = tile.thread_rank();
            result success = FAILED;

            //Perform the insertions
            uint32_t work_queue;
            while (work_queue = tile.ballot(to_insert)) {

                auto cur_lane = __ffs(work_queue) - 1;
                auto cur_k = tile.shfl(k, cur_lane);
                //printf("%i: \tThread Starting Insertion of %" PRIu64 "\n", getThreadID(), cur_k);
                auto cur_result = insertIntoTable(cur_k, T, hashlist, tile);
                if (tile.thread_rank() == cur_lane) {
                    to_insert = false;
                    success = cur_result;
                }
                //printf("%i: \tInsertion Done\n", getThreadID());
            }
            //printf("%i: \tInsertion of  %" PRIu64" result:%i\n", getThreadID(), k, success);
            return success;
        }

        //Public insertion call
        GPUHEADER_D
#ifdef GPUCODE
            result insert(uint64_cu k, bool to_check = true) {
#else
            result insert(uint64_cu k, SpinBarrier * barrier) {
#endif

            return coopInsert(to_check, k);
        };

        //Public Lookup call
        GPUHEADER_D
        bool coopLookup(bool to_lookup, uint64_cu k){
            //printf("%i: Coop Lookup\n", getThreadID());
            //Iterate over hash functions and check if found
            cg::thread_block thb = cg::this_thread_block();
            cg::thread_block_tile<tile_sz> tile = cg::tiled_partition<tile_sz>(thb);
            auto thread_rank = tile.thread_rank();
            bool success = true;
            //Perform the insertions

            uint32_t work_queue;
            while (work_queue = tile.ballot(to_lookup)) {
                auto cur_lane = __ffs(work_queue) - 1;
                auto cur_k = tile.shfl(k, cur_lane);
                auto cur_result = lookup(cur_k, T, tile);

                if (tile.thread_rank() == cur_lane) {
                    to_lookup = false;
                    success = cur_result;
                }
            }
            //printf("%i: key:%" PRIu64 " result:%i\n", getThreadID(), k, success);
            return success;
            //printf("\t\t Lookup Failed\n");
        };

        //Clear all Table Entries
        GPUHEADER
        void clear(){
            for (int i = 0; i < B; i++) {
                for (int j = 0; j < Bs; j++) {
                    new (&T[i*Bs + j]) ClearyCuckooEntryCompact<addtype, remtype>();
                }
            }
        }

        //Get the size of the Table
        GPUHEADER
        int getSize(){
            return tablesize;
        }

        //Return a copy of the hashlist
        GPUHEADER
        int* getHashlistCopy() {
            int* res = new int[hn];
            for (int i = 0; i < hn; i++) {
                res[i] = hashlist[i];
            }
            return res;
        }

        //Transform a vector to a list
        GPUHEADER_H
        std::vector<uint64_cu> toList() {
            std::vector<uint64_cu> list;
            for (int i = 0; i < tablesize; i++) {
                for (int j = 0; j < Bs; j++) {

                    if (T[i].getO(j)) {
                        hashtype h_old = reformKey(i, T[i].getR(j), AS);
                        keytype x = RHASH_INVERSE(HFSIZE_BUCKET, T[i].getH(j), h_old);
                        list.push_back(x);
                    }
                }
            }
            return list;
        }

        //Iterate through all entries and do a read
        void readEverything(int N) {
            int j = 0;
            int step = 1;

            if (N < tablesize) {
                step = std::ceil(((float)tablesize) / ((float)N));
            }

            for (int i = 0; i < tablesize; i+= step) {
                int add = i * Bs + j;

                addtype real_add = (addtype)(add / tile_sz);
                addtype subIndex = (addtype)(add % tile_sz);

                j += T[real_add].getR(subIndex);
            }

            if (j != 0) {
                //printf("Not all Zero\n");
            }
        }


        //Public print call
        GPUHEADER
        void print(){
            //printf("Hashlist:");
            for (int i = 0; i < hn; i++) {
                //printf("%i,", hashlist[i]);
            }
            //printf("\n");
            print(T);
        }

        //Method used for debugging
        GPUHEADER
        void debug(uint64_cu i) {

        }

        //Set the number of rehashes allowed
        void setMaxRehashes(int x){
            MAXREHASHES = x;
        }

        //Set the number of loops allowed
        void setMaxLoops(int x){
            MAXLOOPS = x;
        }

        //Get the number of hashes
        int getHashNum() {
            return hn;
        }

        GPUHEADER
        int getBucketSize() {
            return Bs;
        }

};



//Method to fill ClearyCuckooBucketedtable
template <int tile_sz>
GPUHEADER_G
#ifdef GPUCODE
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, int* failFlag=nullptr, addtype begin = 0, int* count = nullptr, int id = 0, int s = 1)
#else
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, SpinBarrier* barrier, int* failFlag = nullptr, addtype begin = 0, int id = 0, int s = 1)
#endif
{
#ifdef GPUCODE
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    int max = calcBlockSize(N, H->getBucketSize());
    int localCounter = 0;

    //printf("Thread %i Starting - max %i\n", getThreadID(), max);
    for (int i = index + begin; i < max + begin; i += stride) {

        bool realVal = false;
        keytype ins = 0;
        if(i < N + begin){
            realVal = true;
            ins = vals[i];
        }

        //printf("Inserting: %" PRIu64 "\n", ins);

        result res = H->insert(ins, realVal);
        if (res == INSERTED) {
            localCounter++;
        }
        if (res == FAILED) {
            if (failFlag != nullptr && realVal) {
                (*failFlag) = true;
            }
        }
        
    }

    if (count != nullptr) {
        atomicAdd(count, localCounter);
    }
}


#ifdef GPUCODE
//Method to fill ClearyCuckooBucketedtable with a failCheck on every insertion
template <int tile_sz>
GPUHEADER_G
void fillClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz> * H, addtype* occupancy, int* failFlag, int id = 0, int s = 1)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;


    int max = calcBlockSize(N, H->getBucketSize());

    for (int i = index; i < max; i += stride) {
        if (failFlag[0]) {
            break;
        }

        bool realVal = false;
        keytype ins = 0;
        if (i < N) {
            realVal = true;
            ins = vals[i];
        }

        if (H->insert(ins, realVal) == FAILED) {
            if (realVal) {
                atomicCAS(&(failFlag[0]), 0, 1);
            }
        }
        atomicAdd(&occupancy[0], 1);
    }
}
#endif

//Method to check whether a ClearyCuckooBucketed table contains a set of values
template <int tile_sz>
GPUHEADER_G
void checkClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, bool* res, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    int max = calcBlockSize(N, H->getBucketSize());

    for (int i = index; i < max; i += stride) {
        bool realVal = false;
        keytype look = 0;
        if (i < N) {
            realVal = true;
            look = vals[i];
        }

        if (!(H->coopLookup(realVal, look))) {
            res[0] = false;
        }
    }
}

//Method to do lookups in a ClearyCuckooBucketed table on an array of values
template <int tile_sz>
GPUHEADER_G
void lookupClearyCuckooBucketed(int N, int start, int end, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, int id = 0, int s = 1) {
#ifdef GPUCODE
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    int max = calcBlockSize(N, H->getBucketSize());

    for (int i = index; i < max; i += stride) {
        bool realVal = false;
        keytype look = 0;
        if (i < N) {
            realVal = true;
            look = vals[(i + start) % end];
        }
        H->coopLookup(realVal, look);
    }
}

//Method to fill ClearyCuckoo table
template <int tile_sz>
GPUHEADER_G
void dupCheckClearyCuckooBucketed(int N, uint64_cu* vals, ClearyCuckooBucketed<tile_sz>* H, addtype begin = 0)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;

    int max = calcBlockSize(N, H->getBucketSize());

    //printf("Thread %i Starting\n", getThreadID());
    for (int i = index + begin; i < max + begin; i += stride) {
        bool realVal = false;
        keytype k = 0;
        if (i < N + begin) {
            realVal = true;
            k = vals[i];
        }
        H->coopDupCheck(realVal, k);
    }
    //printf("Insertions %i Over\n", getThreadID());
}
